#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <vector>
#include <algorithm>
#include <chrono>
#include <random>
#include <atomic>
#include <stdio.h>

#include <particle.h>


#define NUM_STREAMS 2
#define NOT_FOUND -1

using namespace std;

mt19937 rng;   // random number generator
random_device rd;
__managed__ int n, l, r, s // number of particles, length of side of square, radius of particles, steps


class Particle {
public:
    operator string() const { // overloading the cast operator to convert the object to a string
        char buffer[200];
        snprintf(buffer, 200, "%d %.8lf %.8lf %.8lf %.8lf", i, x, y, vX, vY);
        return buffer;
    }

    // attributes are explicitly declared as public
    int i;
    double x;
    double y;
    double vX;
    double vY;
    int pColl; // particle collisions. 0 = no collision, 1 = collision with particle
    int wColl; // wall collisions. 0 = no collision, 1 = collision with left wall, 2 = collision with right wall, 3 = collision with top wall, 4 = collision with bottom wall

    Particle() {};

    Particle(int i, double x, double y, double vX, double vY, int l) {
        this->i = i;
        this->x = x;
        this->y = y;
        this->vX = vX;
        this->vY = vY;
        this->pColl = 0;
        this->wColl = 0;
    }

    __device__ __host__ int getIndex() {
        return this->i;
    }

    string getFullRepresentation() {
        return [=] {
            std::ostringstream stream;
            stream << std::fixed << std::setprecision(8); // precision of 8 decimal places
            stream << i << " " << x << " " << y << " " << vX << " " << vY << " " << pColl << " " << wColl;
            return stream.str();
        };
    }
};

class CollisionEvent {
public:
    const static int WALL = 0;
    const static int PARTICLE = 1;
    const static int NONE = 2;

    Particle *first_p; // first colliding particle
    Particle *second_p; // second colliding particle
    double time; // time of collision
    int type; // type of collision, refer to the 3 constants above

    __host__ __device__ bool operator==(CollisionEvent other) {
        int firstIndex = (*this->first).getIndex();
        int otherFirstIndex = (*other.first).getIndex();
        if (this->second != nullptr) {
            int secondIndex = (*this->second).getIndex();
            int otherSecondIndex = (*other.second).getIndex();
            return (firstIndex == otherSecondIndex && secondIndex == otherFirstIndex) ||
                   (firstIndex == otherFirstIndex && secondIndex == otherSecondIndex);
        }
        return (firstIndex == otherFirstIndex);
    }


    __host__ __device__ bool operator<(CollisionEvent other) {
        if (this->time == other.getTime()) {
            return this->getSmallestIndex() > other.getSmallestIndex();
        } else {
            return this->time > other.getTime();
        }
    }

    __host__ __device__ CollisionEvent() {}

    __host__ __device__ virtual ~CollisionEvent() {}

    // wall collision event
    __host__ __device__ CollisionEvent(Particle *first, double time) {
        this->first = first;
        this->second = nullptr; // second particle is null for wall collisions
        this->time = time;
        this->type = this->WALL;
    }

    // particle collision event
    __host__ __device__ CollisionEvent(Particle *first, Particle *second, double time) {
        this->first = first;
        this->second = second;
        this->time = time;
        this->type = this->PARTICLE;
    }

    // no collision event
    __host__ __device__ CollisionEvent(Particle *first) {
        this->first = first;
        this->second = nullptr;
        this->time = 1.0;
        this->type = this->NONE;
    }

    __host__ __device__ double getTime() {
        return this->time;
    }

    __host__ __device__ int getType() {
        return this->type;
    }

    __host__ __device__ double getSmallestIndex() {
        if (this->second != nullptr) {
            return (*first).getIndex() < (*second).getIndex() ? (*first).getIndex() : (*second).getIndex();
        }
        return (*first).getIndex();
    }
};


// cuda streams
hipStream_t streams[NUM_STREAMS];

// particle objects
__managed__ Particle *particles; // array of particles

// collision times arrays
// store in a __managed__ array so that it can be accessed by both host and device instead of copying it back and forth
__managed__ double **particleCollisionTimes; // 2D array of particle collision times of n * n size
__managed__ double *wallCollisionTimes; // array of wall collision times of n size

// collision events
__managed__ CollisionEvent *temp; // temporary array to store collision events
__managed__ int *found; // array to store the number of collisions found for each particle

__managed__ CollisionEvent **particleCollisions; // array of particle collision events
__managed__ int particleCollisionsCount; // number of particle collision events

__managed__ CollisionEvent **wallCollisions; // array of wall collision events
__managed__ int wallCollisionsCount; // number of wall collision events

__managed__ CollisionEvent **noCollisions; // array of no collision events
__managed__ int noCollisionsCount; // number of no collision events

// function headers
__host__ void moveParticles(Particle *particles);

__global__ void findEarliestCollision();

__global__ void timeParticleCollision();

__global__ void timeWallCollision();

__global__ void executeParticleCollision();

__global__ void executeWallCollision();

__global__ void executeNoCollision();

__host__ int main(void) { // main is on host
    string command;
    cin >> n >> l >> r >> s >> command;

    // random init
    rng.seed(rd(42));
    uniform_real_distribution<double> pos(r, l - r);
    uniform_real_distribution<double> velocity((double) l / (8 * r), (double) l / 4);

    // assign a device space for the particles array
    hipError_t cudaStatus = hipMallocManaged(&particles, sizeof(Particle) * n);
    int scan_count;
    for (scan_count = 0; scan_count < n; ++scan_count) {
        int index;
        double x;
        double y;
        double vX; // velocity in x direction
        double vY; // velocity in y direction
        int count;
        count = scanf("%d %lf %lf %lf %lf", &index, &x, &y, &vX, &vY);
        if (count == EOF || count <= 0) {
            break;
        }
        particles[scan_count] = Particle(index, x, y, vX, vY, l);
    }
    for (int j = scan_count; j < n; j++) {
        double x = pos(rng);
        double y = pos(rng);
        double vX = velocity(rng);
        double vY = velocity(rng);
        particles[scan_count] = Particle(j, x, y, vX, vY, l);
    }


    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamCreate(&streams[i]);
    }
    hipMallocManaged(&particleCollisionTimes, sizeof(double *) * n);
    for (int i = 0; i < n; ++i) {
        hipMallocManaged(&particleCollisionTimes[i], sizeof(double) * n);
    }
    hipMallocManaged(&wallCollisionTimes, sizeof(double) * n);
    hipMallocManaged(&found, sizeof(int) * n);
    hipMallocManaged(&particleCollisions, sizeof(CollisionEvent *) * n);
    hipMallocManaged((void **) &particleCollisionsCount, sizeof(int));
    hipMallocManaged(&wallCollisions, sizeof(CollisionEvent *) * n);
    hipMallocManaged((void **) &wallCollisionsCount, sizeof(int));
    hipMallocManaged(&noCollisions, sizeof(CollisionEvent *) * n);
    hipMallocManaged((void **) &noCollisionsCount, sizeof(int));
    hipMallocManaged(&temp, sizeof(CollisionEvent) * n);

    // recording start time
    auto start = chrono::high_resolution_clock::now();

    for (int i = 0; i < s; ++i) {
        if (!command.compare("print")) {
            for (int j = 0; j < n; ++j) {
                cout << i << " " << (string) particles[j] << endl;
            }
        }
        moveParticles(particles); // simulation
    }

    // recording end time
    auto finish = chrono::high_resolution_clock::now();

    for (int j = 0; j < n; ++j) {
        cout << s << " " << particles[j].getFullRepresentation() << endl;
    }
    double timeTaken = (double) chrono::duration_cast<chrono::nanoseconds>(finish - start).count() / 1000000000;
    printf("Time taken: %.5f s for %d steps\n", timeTaken, s);
    return 0;
}


__host__ void moveParticles(Particle *particles) {
    for (int i = 0; i < n; ++i) {
        found[i] = NOT_FOUND;
    }
    particleCollisionsCount = 0;
    wallCollisionsCount = 0;
    noCollisionsCount = 0;

    // calculate collision times
    timeWallCollision<<<(n - 1) / 64 + 1, 64, 0, streams[0]>>>();

    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid((n - 1) / 16 + 1, (n - 1) / 16 + 1);
    timeParticleCollision<<<blocksPerGrid, threadsPerBlock, 0, streams[1]>>>();

    hipDeviceSynchronize();

    // find valid collisions
    int foundCount = 0;
    while (foundCount != n) {
        findEarliestCollision<<<(n - 1) / 64 + 1, 64>>>();
        hipDeviceSynchronize();
        for (int i = 0; i < n; ++i) {
            if (found[i] != NOT_FOUND) continue;
            CollisionEvent *e = &temp[i];
            // particle-particle collision
            if ((*e).getType() == CollisionEvent::PARTICLE) {
                int otherIndex = (*(*e).second).getIndex();
                if (temp[otherIndex].getType() == CollisionEvent::PARTICLE) {
                    if ((*e) == temp[otherIndex] && i < otherIndex) {
                        found[i] = 0;
                        found[otherIndex] = 0;
                        foundCount += 2;
                        particleCollisions[particleCollisionsCount++] = e;
                    }
                }
            }
                // particle-wall collision
            else if ((*e).getType() == CollisionEvent::WALL) {
                found[i] = 0;
                ++foundCount;
                wallCollisions[wallCollisionsCount++] = e;
            }
                // no collision
            else {
                found[i] = 0;
                ++foundCount;
                noCollisions[noCollisionsCount++] = e;
            }
        }
    }

    // apply valid collisions
    executeParticleCollision<<<(particleCollisionsCount - 1) / 64 + 1, 64, 0, streams[0]>>>();
    executeWallCollision<<<(wallCollisionsCount - 1) / 64 + 1, 64, 0, streams[1]>>>();
    executeNoCollision<<<(n - 1) / 64 + 1, 64, 0, streams[1]>>>();
    hipDeviceSynchronize();
}

__global__ void findEarliestCollision() {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n || found[index] != NOT_FOUND) return;
    // first assume no collision
    temp[index] = CollisionEvent(&particles[index]);

    // check for particle-wall collision
    if (wallCollisionTimes[index] < temp[index].getTime() && wallCollisionTimes[index] < 1) {
        temp[index] = CollisionEvent(&particles[index], wallCollisionTimes[index]);
    }

    // check for particle-particle collision
    for (int j = 0; j < n; ++j) {
        if (index == j || found[j] != NOT_FOUND) continue;
        double time = particleCollisionTimes[index][j];

        if (time > -1 && time < temp[index].getTime() && time < 1) {
            temp[index] = CollisionEvent(&particles[index], &particles[j], time);
        }
    }

}


// "input": 2 Particles
// "output": Returns time taken before collision occurs if they collide, negative value otherwise.
__global__ void timeParticleCollision() {
    int firstIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int secondIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (firstIndex >= n || secondIndex >= n || firstIndex >= secondIndex) return;
    else {
        Particle first = particles[firstIndex];
        Particle second = particles[secondIndex];
        //a, b and c are as in the quadratic formula representation.
        //t, the time taken for the 2 circles to touch, is the unknown variable we are solving for
        //by taking difference in circle centres, setting an unknown t for collision time, and then taking distance moved in time t,
        //we can solve for t such that the circle centers are <= 2r and therefore collide. 4r^2 is to solve for radius distance.
        double c = pow((first.x - second.x), 2) + pow((first.y - second.y), 2) - 4 * r * r;
        double b = 2 * ((first.x - second.x) * (first.vX - second.vX) + (first.y - second.y) * (first.vY - second.vY));
        double a = pow((first.vX - second.vX), 2) + pow((first.vY - second.vY), 2);
        //check for solution
        double solfirst;
        if (b * b - 4 * a * c < 0) {
            solfirst = 100000.0;
        } else {
            //else if there is a solution, the one with smaller value should be the main collision. Second value is after the 2 circles phase through each other
            solfirst = (-sqrt(b * b - 4 * a * c) - b) / (2 * a);
            if (solfirst < 0) {
                solfirst = (sqrt(b * b - 4 * a * c) - b) / (2 * a);
                if (solfirst > 0) solfirst = 0;
                else solfirst = 100000.0;
            }
        }
        particleCollisionTimes[first.i][second.i] = solfirst;
        particleCollisionTimes[second.i][first.i] = solfirst;
    }
}

// "input": 1 Particle
// "output": Returns time taken before collision occurs if it collides with wall, negative value otherwise.
__global__ void timeWallCollision() {
    int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (particleIndex >= n) {
        return;
    } else {
        Particle particle = particles[particleIndex];
        //check for x wall, y wall collisions
        double xCollide =
                particle.vX < 0 ? (particle.x - r) / (0 - particle.vX) : ((double) l - particle.x - r) / particle.vX;
        double yCollide =
                particle.vY < 0 ? (particle.y - r) / (0 - particle.vY) : ((double) l - particle.y - r) / particle.vY;
        wallCollisionTimes[particle.i] = fmin(xCollide, yCollide);
    }
}

__global__ void executeParticleCollision() {
    int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (particleIndex < particleCollisionsCount) {
        CollisionEvent *e = particleCollisions[particleIndex];
        Particle *first = e->first;
        Particle *second = e->second;
        double time = e->time;
        if (first->getIndex() >= second->getIndex())
            return;
        //move them to proper position first
        first->x += time * first->vX;
        first->y += time * first->vY;
        second->x += time * second->vX;
        second->y += time * second->vY;

        //perform collision here
        //find normal vector
        double normalX = first->x - second->x;
        double normalY = first->y - second->y;
        double normalMag = sqrt(pow(normalX, 2) + pow(normalY, 2));
        normalX = normalX / normalMag;
        normalY = normalY / normalMag;
        double tangentX = -normalY;
        double tangentY = normalX;
        //compute velocity vectors wrt to normal and tangent
        double vFirstNormal = normalX * first->vX + normalY * first->vY;
        double vFirstTangent = tangentX * first->vX + tangentY * first->vY;
        double vSecondNormal = normalX * second->vX + normalY * second->vY;
        double vSecondTangent = tangentX * second->vX + tangentY * second->vY;
        //collision simply swaps velocities
        double temp = vFirstNormal;
        vFirstNormal = vSecondNormal;
        vSecondNormal = temp;
        first->vX = vFirstNormal * normalX + vFirstTangent * tangentX;
        first->vY = vFirstNormal * normalY + vFirstTangent * tangentY;
        second->vX = vSecondNormal * normalX + vSecondTangent * tangentX;
        second->vY = vSecondNormal * normalY + vSecondTangent * tangentY;
        //eliminate negative 0s
        if (first->vX == -0.0) first->vX = 0.0;
        if (first->vY == -0.0) first->vY = 0.0;
        if (second->vX == -0.0) second->vX = 0.0;
        if (second->vY == -0.0) second->vY = 0.0;
        //Continue to move them here
        //Check for wall collisions and stop the particle at wall if so
        double timeToMove;
        double xCollide = first->vX < 0 ? (first->x - r) / (0 - first->vX) : ((double) l - r - first->x) / first->vX;
        double yCollide = first->vY < 0 ? (first->y - r) / (0 - first->vY) : ((double) l - r - first->y) / first->vY;
        if (xCollide >= 1 - time && yCollide >= 1 - time) {
            timeToMove = 1 - time;
        } else {
            timeToMove = fmin(xCollide, yCollide);
        }
        first->x += timeToMove * first->vX;
        first->y += timeToMove * first->vY;

        xCollide = second->vX < 0 ? (second->x - r) / (0 - second->vX) : ((double) l - r - second->x) / second->vX;
        yCollide = second->vY < 0 ? (second->y - r) / (0 - second->vY) : ((double) l - r - second->y) / second->vY;
        if (xCollide >= 1 - time && yCollide >= 1 - time) {
            timeToMove = 1 - time;
        } else {
            timeToMove = fmin(xCollide, yCollide);
        }
        second->x += timeToMove * second->vX;
        second->y += timeToMove * second->vY;
        first->pColl++;
        second->pColl++;
    }
}

__global__ void executeWallCollision() {
    int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (particleIndex < wallCollisionsCount) {
        CollisionEvent *e = wallCollisions[particleIndex];
        Particle *first = e->first;
        //check for x wall collisions
        //check for y wall collisions
        double xCollide = first->vX < 0 ? (first->x - r) / (0 - first->vX) : ((double) l - first->x - r) / first->vX;
        double yCollide = first->vY < 0 ? (first->y - r) / (0 - first->vY) : ((double) l - first->y - r) / first->vY;
        double earlierTime = fmin(xCollide, yCollide);
        double laterTime = fmax(xCollide, yCollide);
        first->x += earlierTime * first->vX;
        first->y += earlierTime * first->vY;
        //Reverse direction depending on which collision happens first
        if (xCollide <= yCollide) {
            first->vX = -first->vX;
        }
        if (yCollide <= xCollide) {
            first->vY = -first->vY;
        }
        //artificially set timing to allow particle to continue after hitting corner
        if (xCollide == yCollide) {
            laterTime = 1;
        }
        first->x += (fmin(1.0, laterTime) - earlierTime) * first->vX;
        first->y += (fmin(1.0, laterTime) - earlierTime) * first->vY;
        first->wColl++;
    }
}

__global__ void executeNoCollision() {
    int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (particleIndex < noCollisionsCount) {
        CollisionEvent *e = noCollisions[particleIndex];
        Particle *first = e->first;
        first->x += first->vX;
        first->y += first->vY;
    }
}
